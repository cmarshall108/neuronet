#include "hip/hip_runtime.h"
#include <neuronet/backends/cuda/cuda_backend.h>
#include <neuronet/core/tensor.h>
#include <neuronet/utils/logging.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace neuronet {
namespace cuda {

// Global cublas handle
static hipblasHandle_t cublas_handle = nullptr;

bool initialize() {
    hipError_t cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        log_error("Failed to set CUDA device: {}", hipGetErrorString(cuda_status));
        return false;
    }

    hipblasStatus_t cublas_status = hipblasCreate(&cublas_handle);
    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        log_error("Failed to create cuBLAS handle");
        return false;
    }

    log_info("CUDA backend initialized successfully");
    return true;
}

void cleanup() {
    if (cublas_handle) {
        hipblasDestroy(cublas_handle);
        cublas_handle = nullptr;
    }
    hipDeviceReset();
}

// CUDA kernel for element-wise addition
__global__ void add_kernel(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA kernel for element-wise multiplication 
__global__ void mul_kernel(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] * b[idx];
    }
}

// CUDA kernel for ReLU activation
__global__ void relu_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] > 0 ? input[idx] : 0;
    }
}

hipblasHandle_t get_cublas_handle() {
    if (!cublas_handle) {
        initialize();
    }
    return cublas_handle;
}

} // namespace cuda
} // namespace neuronet
