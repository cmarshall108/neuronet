#include "hip/hip_runtime.h"
#include <neuronet/backends/cuda/cuda_backend.h>
#include <neuronet/core/tensor.h>
#include <neuronet/utils/logging.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

namespace neuronet {
namespace cuda {

// Global cublas handle
static hipblasHandle_t cublas_handle = nullptr;

// Helper function to determine if the GPU is Tesla K80
bool isTeslaK80() {
    int device_count;
    hipGetDeviceCount(&device_count);
    
    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        // Tesla K80 has compute capability 3.7
        if (prop.major == 3 && prop.minor == 7 && 
            std::string(prop.name).find("Tesla K80") != std::string::npos) {
            return true;
        }
    }
    return false;
}

bool initialize() {
    hipError_t cuda_status = hipSetDevice(0);
    if (cuda_status != hipSuccess) {
        log_error("Failed to set CUDA device: {}", hipGetErrorString(cuda_status));
        return false;
    }

    // Create cuBLAS handle
    hipblasStatus_t cublas_status = hipblasCreate(&cublas_handle);
    if (cublas_status != HIPBLAS_STATUS_SUCCESS) {
        log_error("Failed to create cuBLAS handle");
        return false;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    // Log device information
    log_info("CUDA device: {} (compute capability {}.{})", prop.name, prop.major, prop.minor);
    log_info("Total global memory: {} MB", prop.totalGlobalMem / (1024 * 1024));
    log_info("Multiprocessors: {}", prop.multiProcessorCount);
    
    // Apply Tesla K80 specific optimizations if detected
    if (isTeslaK80()) {
        log_info("Tesla K80 detected, applying specific optimizations");
        
        // For K80, prefer L1 cache over shared memory
        hipDeviceSetCacheConfig(hipFuncCachePreferL1);
        
        // Set cuBLAS math mode to prefer throughput
        hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
    } else {
        log_info("Generic CUDA device, using standard configuration");
    }

    log_info("CUDA backend initialized successfully");
    return true;
}

void cleanup() {
    if (cublas_handle) {
        hipblasDestroy(cublas_handle);
        cublas_handle = nullptr;
    }
    hipDeviceReset();
}

// CUDA kernel for element-wise addition
__global__ void add_kernel(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA kernel for element-wise multiplication 
__global__ void mul_kernel(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] * b[idx];
    }
}

// CUDA kernel for ReLU activation
__global__ void relu_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] > 0 ? input[idx] : 0;
    }
}

hipblasHandle_t get_cublas_handle() {
    if (!cublas_handle) {
        initialize();
    }
    return cublas_handle;
}

// Get CUDA capabilities
void getDeviceCapabilities(int* major, int* minor) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    *major = prop.major;
    *minor = prop.minor;
}

// Get device name
std::string getDeviceName() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    return std::string(prop.name);
}

// Get total device memory in bytes
size_t getTotalMemory() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    return prop.totalGlobalMem;
}

} // namespace cuda
} // namespace neuronet
